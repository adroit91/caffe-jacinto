#include "hip/hip_runtime.h"
#include "caffe/quantized_layer.hpp"
#include "caffe/quantized_layer.cuh"

namespace caffe {


template<typename Dtype>
void QuantizedLayer<Dtype>::Quantize_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  if (this->layer_param_.has_quantization_param()) {
    //LOG(INFO) << "Quantizing layer: " << this->layer_param_.name();
    const vector<shared_ptr<Blob<Dtype> > >& blobs = this->blobs();
    const QuantizationParameter& param = this->layer_param_.quantization_param();
    if (param.precision() != QuantizationParameter_Precision_FLOAT) {
      // Trim layer input
      if(param.quantize_layer_in()) {
        for (int i = 0; i < bottom.size(); ++i) {
          this->QuantizeLayerInputs_gpu(bottom[i]->mutable_gpu_data(), i, bottom[i]->count());
        }
      }

      // Trim weights
      if(param.quantize_layer_weights() && blobs.size() > 0) {
        this->QuantizeWeights_gpu(blobs[0]->mutable_gpu_data(), blobs[0]->count(), param.rounding_scheme(), true);
        if (blobs.size() > 1) { //(this->bias_term_) {
          this->QuantizeWeights_gpu(blobs[1]->mutable_gpu_data(), blobs[1]->count(), param.rounding_scheme(), false);
        }
      }

      // Trim layer output
      if(param.quantize_layer_out()) {
        for (int i = 0; i < top.size(); ++i) {
          this->QuantizeLayerOutputs_gpu(top[i]->mutable_gpu_data(), top[i]->count());
        }
      }
    }
  }
}


template <typename Dtype>
void QuantizedLayer<Dtype>::QuantizeWeights_gpu(Dtype* data, const int count, const int rounding, bool clip) {
  const QuantizationParameter& param = this->layer_param_.quantization_param();
  switch (param.precision()) {
  case QuantizationParameter_Precision_MINIFLOAT:
    Trim2MiniFloat_gpu(data, count, param.mant_bits(), param.exp_bits(), param.rounding_scheme());
    break;
  case QuantizationParameter_Precision_DYNAMIC_FIXED_POINT:
    Trim2FixedPoint_gpu(data, count, param.bw_weights(), param.rounding_scheme(), param.fl_weights(),
    		0, false, clip);
    break;
  case QuantizationParameter_Precision_INTEGER_POWER_OF_2_WEIGHTS:
    Trim2IntegerPowerOf2_gpu(data, count, param.exp_min(), param.exp_max(),
        rounding);
    break;
  case QuantizationParameter_Precision_FLOAT:
	  break;
  default:
    LOG(FATAL) << "Unknown trimming mode: " << param.precision() << " for layer:" << this->layer_param_.name();
    break;
  }
}

template <typename Dtype>
void QuantizedLayer<Dtype>::QuantizeLayerInputs_gpu(
    Dtype* data, const int blob_id, const int count) {
  const QuantizationParameter& param = this->layer_param_.quantization_param();
  bool unsigned_layer_in = param.unsigned_layer_in_size()>0? param.unsigned_layer_in(blob_id): false;
  switch (param.precision()) {
    case QuantizationParameter_Precision_INTEGER_POWER_OF_2_WEIGHTS:
    case QuantizationParameter_Precision_DYNAMIC_FIXED_POINT:
      if(param.fl_layer_in_size() > blob_id) {
        Trim2FixedPoint_gpu(data, count, param.bw_layer_in(), param.rounding_scheme(), param.fl_layer_in(blob_id),
    		  0, unsigned_layer_in, true);
      }
      break;
    case QuantizationParameter_Precision_MINIFLOAT:
      Trim2MiniFloat_gpu(data, count, param.mant_bits(), param.exp_bits(), param.rounding_scheme());
      break;
    case QuantizationParameter_Precision_FLOAT:
  	  break;
    default:
      LOG(FATAL) << "Unknown trimming mode: " << param.precision() << " for layer:" << this->layer_param_.name();
      break;
  }
}

template <typename Dtype>
void QuantizedLayer<Dtype>::QuantizeLayerOutputs_gpu(Dtype* data,
      const int count) {
  const QuantizationParameter& param = this->layer_param_.quantization_param();
  switch (param.precision()) {
    case QuantizationParameter_Precision_INTEGER_POWER_OF_2_WEIGHTS:
    case QuantizationParameter_Precision_DYNAMIC_FIXED_POINT:
      Trim2FixedPoint_gpu(data, count, param.bw_layer_out(), param.rounding_scheme(), param.fl_layer_out(),
    		  0, param.unsigned_layer_out(), true);
      break;
    case QuantizationParameter_Precision_MINIFLOAT:
      Trim2MiniFloat_gpu(data, count, param.mant_bits(), param.exp_bits(), param.rounding_scheme());
      break;
    case QuantizationParameter_Precision_FLOAT:
  	  break;
    default:
      LOG(FATAL) << "Unknown trimming mode: " << param.precision() << " for layer:" << this->layer_param_.name();
      break;
  }
}

template <typename Dtype>
__global__ void Trim2FixedPoint_kernel(Dtype* data, const int cnt,
      const int bit_width, const int rounding, const int fl, Dtype sparsity_threshold, bool unsigned_data, bool clip) {
	CUDA_KERNEL_LOOP(index, cnt) {
    
    data[index] = data[index] * powf(2, fl);

    // Round data
    switch (rounding) {
    case QuantizationParameter_Rounding_NEAREST:
      data[index] = rint(data[index]);
      break;
    case QuantizationParameter_Rounding_STOCHASTIC:
      data[index] = __float2int_rd(data[index] + RandUniform_device(index));
      break;
    default:
      break;
    }

#if CLIP_QUANT
    if(clip) {
    	// Saturate data
    	int qrange = unsigned_data? bit_width :  (bit_width - 1);
    	Dtype max_data = +(powf(2, qrange) - 1);
    	Dtype min_data = unsigned_data? 0 : -(powf(2, qrange));
    	data[index] = (data[index]>max_data?max_data:(data[index]<min_data?min_data:data[index]));
    }
#endif

    if(sparsity_threshold != 0 && fabs(data[index]) < sparsity_threshold) {
    	data[index] = 0;
    }

    data[index] = data[index] * powf(2, -fl);
  }
}

template <typename Dtype>
void QuantizedLayer<Dtype>::Trim2FixedPoint_gpu(Dtype* data, const int cnt,
      const int bit_width, const int rounding, int fl, Dtype sparsity_threshold, bool unsigned_data, bool clip) {
  Trim2FixedPoint_kernel<<<CAFFE_GET_BLOCKS(cnt), CAFFE_CUDA_NUM_THREADS>>>(
      data, cnt, bit_width, rounding, fl, sparsity_threshold, unsigned_data, clip);
}

template <typename Dtype>
__global__ void Trim2MiniFloat_kernel(Dtype* data, const int cnt,
      const int bw_mant, const int bw_exp, const int rounding){
	CUDA_KERNEL_LOOP(index, cnt) {
    Trim2MiniFloat_device(&data[index], bw_mant, bw_exp, rounding, index);
	}
}

template <typename Dtype>
void QuantizedLayer<Dtype>::Trim2MiniFloat_gpu(Dtype* data,
      const int cnt, const int bw_mant, const int bw_exp, const int rounding) {
  Trim2MiniFloat_kernel<<<CAFFE_GET_BLOCKS(cnt), CAFFE_CUDA_NUM_THREADS>>>(
      data, cnt, bw_mant, bw_exp, rounding);
}

template <typename Dtype>
__global__ void Trim2IntegerPowerOf2_kernel(Dtype* data, const int cnt,
      const int min_exp, const int max_exp, const int rounding) {
	CUDA_KERNEL_LOOP(index, cnt) {
    float exponent = log2f(fabs((float)data[index]));
    int sign = data[index] >= 0 ? 1 : -1;
    switch (rounding) {
    case QuantizationParameter_Rounding_NEAREST:
      exponent = rint(exponent);
      break;
    case QuantizationParameter_Rounding_STOCHASTIC:
      exponent = __float2int_rd(exponent + RandUniform_device(index));
      break;
    default:
      break;
    }
    exponent = fmaxf(fminf(exponent, max_exp), min_exp);
    data[index] = sign * powf(2, exponent);
	}
}

template <typename Dtype>
void QuantizedLayer<Dtype>::Trim2IntegerPowerOf2_gpu(Dtype* data,
      const int cnt, const int min_exp, const int max_exp, const int rounding) {
  Trim2IntegerPowerOf2_kernel<<<CAFFE_GET_BLOCKS(cnt), CAFFE_CUDA_NUM_THREADS>>>(
      data, cnt, min_exp, max_exp, rounding);
}



template void QuantizedLayer<float>::Quantize_gpu(const vector<Blob<float>*>& bottom, const vector<Blob<float>*>& top);
template void QuantizedLayer<double>::Quantize_gpu(const vector<Blob<double>*>& bottom, const vector<Blob<double>*>& top);

}  // namespace caffe


